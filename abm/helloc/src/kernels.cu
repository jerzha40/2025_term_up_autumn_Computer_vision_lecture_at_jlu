#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <hip/hip_runtime.h>
#include <cstddef>

__global__ void vecAddKernel(const float *A, const float *B, float *C, size_t n)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    for (size_t idx = i; idx < n; idx += blockDim.x * gridDim.x)
    {
        C[idx] = A[idx] + B[idx];
    }
}
void vector_add_gpu(const float *hA, const float *hB, float *hC, size_t n)
{
    float *dA = nullptr, *dB = nullptr, *dC = nullptr;
    hipMalloc(&dA, n * sizeof(float));
    hipMalloc(&dB, n * sizeof(float));
    hipMalloc(&dC, n * sizeof(float));
    hipMemcpy(dA, hA, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, n * sizeof(float), hipMemcpyHostToDevice);
    int block = 256;
    int grid = static_cast<int>((n + block - 1) / block);
    if (grid > 65535)
        grid = 65535;
    vecAddKernel<<<grid, block>>>(dA, dB, dC, n);
    hipDeviceSynchronize();
    hipMemcpy(hC, dC, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}