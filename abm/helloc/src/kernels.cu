#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <hip/hip_runtime.h>
#include <cstddef>
#include <stdio.h>

__global__ void vecAddKernel(const float *A, const float *B, float *C, size_t n)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    for (size_t idx = i; idx < n; idx += blockDim.x * gridDim.x)
    {
        C[idx] = A[idx] + B[idx];
    }
}

void vector_add_gpu(const float *hA, const float *hB, float *hC, size_t n)
{
    size_t freeB = 0, totalB = 0;
    hipMemGetInfo(&freeB, &totalB);
    printf("VRAM free %.1f GB / total %.1f GB\n", freeB / 1e9, totalB / 1e9);

    hipEvent_t e0, e1, e2, e3;
    hipEventCreate(&e0);
    hipEventCreate(&e1);
    hipEventCreate(&e2);
    hipEventCreate(&e3);

    float *dA = nullptr, *dB = nullptr, *dC = nullptr;
    hipHostRegister((void *)hA, n * sizeof(float), hipHostRegisterDefault);
    hipHostRegister((void *)hB, n * sizeof(float), hipHostRegisterDefault);
    hipHostRegister((void *)hC, n * sizeof(float), hipHostRegisterDefault);
    hipMalloc(&dA, n * sizeof(float));
    hipMalloc(&dB, n * sizeof(float));
    hipMalloc(&dC, n * sizeof(float));
    hipEventRecord(e0);
    hipMemcpy(dA, hA, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, n * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(e1);
    int block = 256;
    int grid = static_cast<int>((n + block - 1) / block);
    if (grid > 65535)
        grid = 65535;
    vecAddKernel<<<grid, block>>>(dA, dB, dC, n);
    hipEventRecord(e2);
    hipDeviceSynchronize();
    hipMemcpy(hC, dC, n * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(e3);
    hipEventSynchronize(e3);
    float tH2D = 0, tK = 0, tD2H = 0;
    hipEventElapsedTime(&tH2D, e0, e1);
    hipEventElapsedTime(&tK, e1, e2);
    hipEventElapsedTime(&tD2H, e2, e3);

    printf("breakdown  H2D: %.3f ms, kernel: %.3f ms, D2H: %.3f ms\n", tH2D, tK, tD2H);

    hipEventDestroy(e0);
    hipEventDestroy(e1);
    hipEventDestroy(e2);
    hipEventDestroy(e3);
    hipHostUnregister((void *)hA);
    hipHostUnregister((void *)hB);
    hipHostUnregister((void *)hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}
void cuda_warmup()
{
    hipFree(0);
}
void vecadd_alloc(float *&dA, float *&dB, float *&dC, size_t n)
{
    hipMalloc(&dA, n * sizeof(float));
    hipMalloc(&dB, n * sizeof(float));
    hipMalloc(&dC, n * sizeof(float));
}
void vecadd_run(const float *dA, const float *dB, float *dC, size_t n)
{
    int block = 256;
    int grid = static_cast<int>((n + block - 1) / block);
    int maxGridX = 0;
    hipDeviceGetAttribute(&maxGridX, hipDeviceAttributeMaxGridDimX, 0);
    grid = (grid > maxGridX ? maxGridX : grid);
    vecAddKernel<<<grid, block>>>(dA, dB, dC, n);
    hipDeviceSynchronize();
}
void vecadd_free(float *dA, float *dB, float *dC)
{
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}
