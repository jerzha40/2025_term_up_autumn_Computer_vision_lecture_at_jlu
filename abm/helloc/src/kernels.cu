#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <hip/hip_runtime.h>
#include <cstddef>
#include <stdio.h>

__global__ void vecAddKernel(const float *A, const float *B, float *C, size_t n)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    for (size_t idx = i; idx < n; idx += blockDim.x * gridDim.x)
    {
        C[idx] = A[idx] + B[idx];
    }
}
void vector_add_gpu(const float *hA, const float *hB, float *hC, size_t n)
{
    hipEvent_t e0, e1, e2, e3;
    hipEventCreate(&e0);
    hipEventCreate(&e1);
    hipEventCreate(&e2);
    hipEventCreate(&e3);

    float *dA = nullptr, *dB = nullptr, *dC = nullptr;
    hipHostRegister((void *)hA, n * sizeof(float), hipHostRegisterDefault);
    hipHostRegister((void *)hB, n * sizeof(float), hipHostRegisterDefault);
    hipHostRegister((void *)hC, n * sizeof(float), hipHostRegisterDefault);
    hipMalloc(&dA, n * sizeof(float));
    hipMalloc(&dB, n * sizeof(float));
    hipMalloc(&dC, n * sizeof(float));
    hipEventRecord(e0);
    hipMemcpy(dA, hA, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, n * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(e1);
    int block = 256;
    int grid = static_cast<int>((n + block - 1) / block);
    if (grid > 65535)
        grid = 65535;
    vecAddKernel<<<grid, block>>>(dA, dB, dC, n);
    hipEventRecord(e2);
    hipDeviceSynchronize();
    hipMemcpy(hC, dC, n * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(e3);
    hipEventSynchronize(e3);
    float tH2D = 0, tK = 0, tD2H = 0;
    hipEventElapsedTime(&tH2D, e0, e1);
    hipEventElapsedTime(&tK, e1, e2);
    hipEventElapsedTime(&tD2H, e2, e3);

    printf("breakdown  H2D: %.3f ms, kernel: %.3f ms, D2H: %.3f ms\n", tH2D, tK, tD2H);

    hipEventDestroy(e0);
    hipEventDestroy(e1);
    hipEventDestroy(e2);
    hipEventDestroy(e3);
    hipHostUnregister((void *)hA);
    hipHostUnregister((void *)hB);
    hipHostUnregister((void *)hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}
void cuda_warmup()
{
    hipFree(0);
}
