#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <algorithm>
#include <kernels.cuh>

__global__ void vecAddKernel(const float *A, const float *B, float *C, size_t n)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    for (size_t idx = i; idx < n; idx += blockDim.x * gridDim.x)
    {
        C[idx] = A[idx] + B[idx];
    }
}

static inline void ck(hipError_t e, const char *msg)
{
    if (e != hipSuccess)
        throw std::runtime_error(std::string(msg) + ": " + hipGetErrorString(e));
}

void vecadd_gpu(const float *hA, const float *hB, float *hC, size_t n)
{
    float *dA = nullptr, *dB = nullptr, *dC = nullptr;
    ck(hipMalloc(&dA, n * sizeof(float)), "hipMalloc dA");
    ck(hipMalloc(&dB, n * sizeof(float)), "hipMalloc dB");
    ck(hipMalloc(&dC, n * sizeof(float)), "hipMalloc dC");
    ck(hipMemcpy(dA, hA, n * sizeof(float), hipMemcpyHostToDevice), "H2D A");
    ck(hipMemcpy(dB, hB, n * sizeof(float), hipMemcpyHostToDevice), "H2D B");

    int block = 256;
    int grid = int((n + block - 1) / block);
    int maxGridX = 0;
    hipDeviceGetAttribute(&maxGridX, hipDeviceAttributeMaxGridDimX, 0);
    grid = std::min(grid, maxGridX);

    vecAddKernel<<<grid, block>>>(dA, dB, dC, n);
    ck(hipGetLastError(), "kernel");
    ck(hipMemcpy(hC, dC, n * sizeof(float), hipMemcpyDeviceToHost), "D2H C");

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}

void vecadd_gpu_chunked(const float *hA, const float *hB, float *hC, size_t n)
{
    size_t freeB = 0, totalB = 0;
    hipMemGetInfo(&freeB, &totalB);
    size_t maxElems = std::max<size_t>(1, (freeB * 8 / 10) / (3 * sizeof(float)));
    float *dA[2]{}, *dB[2]{}, *dC[2]{};
    hipStream_t s[2];
    hipStreamCreate(&s[0]);
    hipStreamCreate(&s[1]);
    hipMalloc(&dA[0], maxElems * sizeof(float));
    hipMalloc(&dB[0], maxElems * sizeof(float));
    hipMalloc(&dC[0], maxElems * sizeof(float));
    hipMalloc(&dA[1], maxElems * sizeof(float));
    hipMalloc(&dB[1], maxElems * sizeof(float));
    hipMalloc(&dC[1], maxElems * sizeof(float));

    size_t chunk = 0;
    for (size_t off = 0; off < n; off += maxElems, ++chunk)
    {
        int p = int(chunk & 1);
        if (chunk >= 2)
            hipStreamSynchronize(s[p]);
        size_t m = std::min(maxElems, n - off);
        hipMemcpyAsync(dA[p], hA + off, m * sizeof(float), hipMemcpyHostToDevice, s[p]);
        hipMemcpyAsync(dB[p], hB + off, m * sizeof(float), hipMemcpyHostToDevice, s[p]);

        int block = 512;
        int grid = int((m + block - 1) / block);
        int maxGridX = 0;
        hipDeviceGetAttribute(&maxGridX, hipDeviceAttributeMaxGridDimX, 0);
        grid = std::min(grid, maxGridX);
        vecAddKernel<<<grid, block, 0, s[p]>>>(dA[p], dB[p], dC[p], m);

        hipMemcpyAsync(hC + off, dC[p], m * sizeof(float), hipMemcpyDeviceToHost, s[p]);
    }
    hipStreamSynchronize(s[0]);
    hipStreamSynchronize(s[1]);
    hipStreamDestroy(s[0]);
    hipStreamDestroy(s[1]);
    hipFree(dA[0]);
    hipFree(dB[0]);
    hipFree(dC[0]);
    hipFree(dA[1]);
    hipFree(dB[1]);
    hipFree(dC[1]);
}
